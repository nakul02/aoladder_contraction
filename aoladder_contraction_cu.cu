#include "hip/hip_runtime.h"
//============================================================================
// Name        : aoladder_contraction_cu.cu
// For declaration similar to data[a1][a0], extents was defined as {a0,a1}.
// This is with respect to the last test code received
// If this definition of extents is changed to {a1,a0}, interchange the use 
// of extents[0] and extents[1].
//============================================================================

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "sip_interface.h"
#include "gpu_super_instructions.h"
#include <time.h>

#define PZP //Enable for PZP and LEC+PZP
//#define LEC//Enable for LEC and LEC+PZP

//CUDA block dimensions
#define BLOCK_SIZE 21

//define this to debug
//#define DEBUG

// Define this to turn on sparsity check
#define SPARSITY_CHECK


//Get the sub-matrix position
__device__ double* GetSubMatrix(double* M, int b_row, int b_col, int stride) {
	return &M[b_col * stride * BLOCK_SIZE + b_row * BLOCK_SIZE];
}

//Get an element from a sub-matrix
__device__ double GetElement(double* M, int block_x, int block_y, int row, int col, int stride_col, int stride_row) {
	return M[col * stride_row + row];
}

//Set result value in global memory
__device__ void SetElement(double* M, int row, int col, double value, int stride) {
	M[col * stride + row] = value;
}

// Block partitioned CUDA implementation
__global__ void contract_part(double* x1, double* x2, double* y,
	 const int* d_extents0, int* d_extents1, int* d_extents2, int b_rows, int b_cols) {
	//calculate the position of sub-block 
	int b_row = blockIdx.x % b_rows;
	int b_col = blockIdx.x / b_rows;

	//Retrieve the sub-block 
	double* y_sub = GetSubMatrix (y, b_row, b_col, d_extents2[1]);

	// Each thread computes one element of y_sub by accumulating results into y_value
    double y_value = 0.0;
	
    // Row no. and Column no. of the element the thread is designated to access
    int row = threadIdx.x % BLOCK_SIZE;
    int col = threadIdx.x / BLOCK_SIZE;

	//Calculate the number of blocks in x1 along the direction to be reduced
	int r_blocks = ceilf((float)d_extents0[0] / (float)BLOCK_SIZE);
	
    // Loop over all the sub-matrices of x1 and x2 that are required to compute y_sub
    // Multiply each pair of sub-matrices together and accumulate the results
    for (int m = 0; m < r_blocks; ++m) {

        // Get sub-matrix Asub of x1 
        double* x1_sub = GetSubMatrix (x1, b_row, m, d_extents0[1]);

        // Get sub-matrix Bsub of x2 
        double* x2_sub = GetSubMatrix (x2, m, b_col, d_extents1[1]);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ double x1s[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double x2s[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ double x1s_sum[BLOCK_SIZE];
	
        // Load x1_sub from device memory to shared memory Each thread loads one element of the sub-matrix
		#ifdef PZP
		// If thread is designated to access a element out of the extent boundaries(overflow)
		// then use zero instead of the element in memory (zero pad).
		if(m * BLOCK_SIZE + col >= d_extents0[0])
			x1s[row][col] = 0.0;
		else
		#endif
        	x1s[row][col] = GetElement(x1_sub, b_col, b_row, row, col, d_extents0[0], d_extents0[1]);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

/****************************** Sparsity Check START **********************************/
#ifdef SPARSITY_CHECK
		// Perform a reduction sum and check if this sub-block is sparse
		// Each thread in row 0 will collect sums of its corresponding columns and then the row will be summed up in x1s_sum[0]
		if (row == 0) {
			x1s_sum[col] = 0.0;
			for (int i = 0; i < BLOCK_SIZE; i++)
				x1s_sum[col] += x1s[i][col];
		}
		__syncthreads();

		if ((row || col) == 0) {
			for (int i = 1; i < BLOCK_SIZE; i++)
				x1s_sum[0] += x1s_sum[i];
		}
		__syncthreads();
		if (x1s_sum[0] == 0)
			continue;
#endif
/****************************** Sparsity Check END **********************************/

        // Load x2_sub from device memory to shared memory
        // Each thread loads one element of the sub-matrix 
#ifdef PZP
		if(m * BLOCK_SIZE + row >= d_extents1[1])
		  	x2s[row][col] = 0.0;
		else
#endif
	      	x2s[row][col] = GetElement(x2_sub,b_col, b_row, row, col, d_extents1[0], d_extents1[1]);

		__syncthreads();
		
	 	int emax = BLOCK_SIZE;
#ifdef LEC
		// Contain leakage to avoid multiplying unnecessary values
		
		// Position of sub-blocks
		int x1_xpos = m * BLOCK_SIZE + BLOCK_SIZE;
		int x2_ypos = m * BLOCK_SIZE + BLOCK_SIZE;
		
		// Calculating the length of sub-block to iterate over
		int e1 = x1_xpos - d_extents0[0]  ;
	
		if (e1 > 0)
			emax -= e1;
#endif		 
		// Multiply x1_sub and x2_sub together
	    for (int e = 0; e < emax; ++e)
	        y_value += x1s[row][e] * x2s[e][col];
	
	    // Synchronize to make sure that the preceding
	    // computation is done before loading two new
	    // sub-matrices of A and B in the next iteration
	    __syncthreads();
    }

    // Write y_sub to device memory
    // Each correctly mapper thread writes one element
	if (((b_row * BLOCK_SIZE + row) < d_extents2[1]) && ((b_col * BLOCK_SIZE + col) < d_extents2[0])) 
		SetElement(y_sub, row, col, y_value, d_extents2[1]);
}


#ifdef __cplusplus
extern "C" {
#endif

void aoladder_contraction_cu_nosparse(
	int& array_slot_0, int& rank_0, int *index_values_0, int& size_0, int *extents_0, double *data_0,
	int& array_slot_1, int& rank_1, int *index_values_1, int& size_1, int *extents_1, double *data_1,
	int& array_slot_2, int& rank_2, int *index_values_2, int& size_2, int *extents_2, double *data_2,
	int& ierr, int blockNum) {

	int devid;
	int rank = 0;
	double* p_y[blockNum];
	double* p_x1[blockNum];
	double* p_x2[blockNum];
	int label_py[]={2,5,4,6};
	int label_x1[]={1,2,3,4};
	int label_x2[]={1,5,3,6};

	//Initialize GPU
	_init_gpu(&devid, &rank);

	//Allocate memory on GPU
      for(int bi=0;bi<blockNum;bi++){
	p_y[bi] = _gpu_allocate (size_2);
	p_x1[bi] = _gpu_allocate (size_0);
	p_x2[bi] = _gpu_allocate (size_1);

	//Transfer blocks to the GPU	
	_gpu_host_to_device(&data_0[bi*size_0], p_x1[bi], size_0);
	_gpu_host_to_device(&data_1[bi*size_1], p_x2[bi], size_1);
	}
	//Perform the contraction operation on GPU

      for(int bi=0;bi<blockNum;bi++){
	_gpu_contract(p_y[bi], rank_2, extents_2, &label_py[0],
		p_x1[bi], rank_0, extents_0, &label_x1[0], p_x2[bi], rank_1,
		extents_1, &label_x2[0]);
}       
 //temporary 
//	_gpu_contract(p_y, rank_2, extents_2, &label_py[0],
//		p_x1, rank_0, extents_0, &label_x1[0], p_x2, rank_1,
//		extents_1, &label_x2[0]);

 //temporary 
//	_gpu_contract(p_y, rank_2, extents_2, &label_py[0],
//		p_x1, rank_0, extents_0, &label_x1[0], p_x2, rank_1,
//		extents_1, &label_x2[0]);

      for(int bi=0;bi<blockNum;bi++){
	//Get results from GPU to CPU
	_gpu_device_to_host(&data_2[bi*size_2], p_y[bi], size_2);
	
	//De-allocate memory on GPU
	_gpu_free(p_y[bi]);
	_gpu_free(p_x1[bi]);
	_gpu_free(p_x2[bi]);
}
}
void compute_aoladder_with_sparsity_gpu(
		int a1, int a2, int b1, int b2,
		double* integrals, int i1, int j1, int size_0, int rank_0, int* extents_0,
		double* t2old, int ii1, int jj1, int size_1, int rank_1, int* extents_1,
		double* t2new, int size_2, int rank_2, int* extents_2) {


/************************************ GPU START *************************************************/
	double* d_y;
	double* d_x1;
	double* d_x2;
	int* d_extents0;
	int* d_extents1;
	int* d_extents2;
	double* check_result = (double*) malloc(sizeof(double) * size_2);

	//Labels Needed in the original CPU version
/*
	int label_py[]={2,5,4,6};
	int label_x1[]={1,2,3,4};
	int label_x2[]={1,5,3,6};
*/
	float time,timeTransfer;
	hipEvent_t start, stop, startTransfer, stopTransfer;

	//Call and time the CUDA kernel
	hipEventCreate(&startTransfer);
	hipEventCreate(&stopTransfer);
	hipEventRecord(startTransfer, 0);


	//Allocate memory on GPU
	hipMalloc ((void **) &d_x1, size_0 * sizeof(double));
	hipMalloc ((void **) &d_x2, size_1 * sizeof(double));
	hipMalloc ((void **) &d_y, size_2 * sizeof(double));
	hipMalloc ((void **) &d_extents0, rank_0 * sizeof(int));
	hipMalloc ((void **) &d_extents1, rank_1 * sizeof(int));
	hipMalloc ((void **) &d_extents2, rank_2 * sizeof(int));

	//Copy the data to GPU	
	hipMemcpy (d_x1, integrals, size_0 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy (d_x2, t2old, size_1 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy (d_extents0, extents_0, rank_0 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy (d_extents1, extents_1, rank_1 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy (d_extents2, extents_2, rank_2 * sizeof(int), hipMemcpyHostToDevice);

	
	//Calculate the block dimensions (one dimension less than that of the matrices)
	//Total threads per block 
	int block_dim_x = BLOCK_SIZE * BLOCK_SIZE;

	//Calculate the grid dimensions 
	int grid_dim_x = ceil((double)extents_2[0] / (double)BLOCK_SIZE) * ceil((double)extents_2[1] / (double)BLOCK_SIZE);
	int block_rows = ceil((double)extents_2[1] / (double)BLOCK_SIZE);
	int block_cols = ceil((double)extents_2[0] / (double)BLOCK_SIZE);

#ifdef DEBUG
	printf("block rows: %d\n", block_rows);
	printf("block cols: %d\n", block_cols);
	printf("block_dim_x = %d\n", block_dim_x);
	printf("grid_dim_x = %d\n", grid_dim_x);
	int r_blocks = ceil(extents_0[0] / BLOCK_SIZE);
	printf("Number of blocks in contraction (r_blocks) = %d\n", r_blocks );
#endif

	//Create dim3 objects
	dim3 dimBlock(block_dim_x);
	dim3 dimGrid(grid_dim_x);

	//Call and time the CUDA kernel
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	contract_part<<<dimGrid, dimBlock>>>(d_x1, d_x2, d_y, d_extents0, d_extents1, d_extents2, block_rows, block_cols);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//Get results from GPU to CPU
	hipMemcpy (t2new, d_y, size_2 * sizeof(double), hipMemcpyDeviceToHost);

#ifdef DEBUG
	int j;
	printf("t2old :\n\n");
	for (i = 0; i < extents_1[1]; i++) {
		for (j = 0; j < extents_1[0]; j++) {
			int index = j * extents_1[1] + i;
			printf("%g " , t2old[index]);
		}
		printf("\n");
	}

	printf("integrals :\n\n");
	for (i = 0; i < extents_0[1]; i++) {
		for (j = 0; j < extents_0[0]; j++) {
			int index = j * extents_0[1] + i;
			printf("%g " , integrals[index]);
		}
		printf("\n");
	}

	printf("t2new (gpu block partition):\n\n");
	for (i = 0; i < extents_2[1]; i++) {
		for (j = 0; j < extents_2[0]; j++) {
			int index = j * extents_2[1] + i;
			printf("%g " , t2new[index]);
		}
		printf("\n");
	}
#endif
					
	//Free the memory up on gpu
	hipFree(d_y);
	hipFree(d_x1);
	hipFree(d_x2);
	
	hipEventRecord(stopTransfer, 0);
	hipEventSynchronize(stopTransfer);
	hipEventElapsedTime(&timeTransfer, startTransfer, stopTransfer);
	std::cout.precision(10);
	std::cout<<"Time taken for contraction is "<<time<<" msec"<<std::endl;
	std::cout<<"Time taken for contraction with data transfer time is "<<timeTransfer<<" msec"<<std::endl;


/************************************ GPU END *************************************************/

}


void aoladder_contraction_cu(
	int& array_slot_0, int& rank_0, int *index_values_0, int& size_0, int *extents_0, double *data_0,
	int& array_slot_1, int& rank_1, int *index_values_1, int& size_1, int *extents_1, double *data_1,
	int& array_slot_2, int& rank_2, int *index_values_2, int& size_2, int *extents_2, double *data_2,
	int& ierr) {

    //offsets of atomi//indeces
    int offset_1, offset_2;
    int one;

    // aosegments
    //     integer(C_INT) n_ao_segs(1), aosegs(6)
    int* pn_ao_segs;
    int* paosegs;


    one = 0; // 1
    ierr = 0;

    // Get the predefined data

    // ----------------------------------------------------------------------
    // Get the ao segment ranges
    predefined_int_array("ao_seg_ranges", one, &pn_ao_segs, &paosegs);
    //     write(6,*) ' NAO_SEGS', pn_ao_segs(1), (paosegs(i), i=1,
    //    *                                             pn_ao_segs(1))
    // ----------------------------------------------------------------------


    // Set offsets for the integral array. These also set the offsets for the
    // ao indices of the amplitude arrays.
    offset_1 = 0;
    offset_2 = 0;

    if (index_values_0[0] > 1)
        offset_1 = paosegs[index_values_0[0] - 1];
    if (index_values_0[1] > 1)
        offset_2 = paosegs[index_values_0[1] - 1];
	
	
	//Call code to contract 2D matrices using block partitioning
    compute_aoladder_with_sparsity_gpu(offset_1, extents_0[0] + offset_1, offset_2,
            extents_0[1] + offset_2,
            data_0, //      integrals data_0
            extents_1[0], extents_1[1], size_0, rank_0, extents_0,
            data_1, //old amplitudes data_1
            extents_2[0], extents_2[1], size_1, rank_1, extents_1,
            data_2, size_2, rank_2, extents_2); //new amplitudes data_2



}
#ifdef __cplusplus
}
#endif
 	
