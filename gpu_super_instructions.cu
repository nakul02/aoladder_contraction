#include "hip/hip_runtime.h"
#ifndef __GPU_SUPER_INSTRUCTIONS_CU__
#define __GPU_SUPER_INSTRUCTIONS_CU__

#include "gpu_super_instructions.h"
#include "sip_interface.h"
#include <hipblas.h>
#include "cuda_check.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

// Internal helper functions
void __gpu_contract_helper(double* y, int ny, int* yDims, int* yInds,
		double* x1, int n1, int* x1Dims, int* x1Inds, double* x2, int n2,
		int* x2Dims, int* x2Inds);
void __gpu_permute_helper(double* y, int ny, int* yDims, int* yInds, double* x1,
		int n1, int* x1Dims, int* x1Inds);

void __gpu_matplus_helper(double* p_y, double* p_x1, double* p_x2, int numElems,
		const double alpha);

void printHostArray(double*, int);
void printGPUArray(double*, int);

__constant__ int dimsDev[MAX_RANK];
__constant__ int stepsDev[MAX_RANK];

hipblasHandle_t cublasHandle;

/**
 * Initialization routine for CUDA
 * @param [out] device id (not used)
 */
void _init_gpu(int* devid, int* myRank) {
	int devCnt = 0;
	hipDeviceProp_t deviceProp;

	hipError_t err = hipGetDeviceCount(&devCnt);

	int myDevice = -1;
	//CUDA_CHECK(hipGetDevice(&myDevice));

	if (err == hipSuccess) {
		//if (myDevice < 0 || myDevice > devCnt){
		//    printf ("Task %d : hipGetDevice did not return a device (device id = %d)\n", *myRank,myDevice);
		myDevice = (*myRank) % devCnt;
		CUDA_CHECK(hipSetDevice(myDevice));
		*devid = myDevice;
		//}
		
		//printf("Task %d set device %d out of %d GPUs\n", *myRank, myDevice,
		//		devCnt);
		//CUDA_CHECK(hipGetDeviceProperties(&deviceProp, myDevice));
		//printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
		//		myDevice, deviceProp.name, deviceProp.major, deviceProp.minor);
		CUBLAS_CHECK(hipblasCreate(&cublasHandle));
	} else {
		*devid = -1;
		printf("Task %d not using GPUs, error returned :%s\n", *myRank,
				hipGetErrorString(err));
	}

}

/**
 * Any cleanup that needs to be done on the GPU
 * is done here.
 */
void _finalize_gpu() {
//    CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

///**
// * Implements Y = X1 + X2.
// * Does either Y+=X1 or Y+=X2
// * @param y         address of y (passed as double pointer since passed in from fortran)
// * @param x1        address of x1
// * @param x2        address of x2
// * @param numElems  number of elements
// */
//void _gpu_matplus(double* y, double* x1, double* x2, int numElems) {
//	double alpha = 1.0;
//	__gpu_matplus_helper(y, x1, x2, numElems, alpha);
//}
//
///**
// * Implements Y = X1 - X2.
// * To do Y-=X, pass  y=Y, x1=x, x2=y
// * @param y         address of y (passed as double pointer since passed in from fortran)
// * @param x1        address of x1
// * @param x2        address of x2
// * @param numElems  number of elements
// */
//void _gpu_matminus(double* y, double* x1, double* x2, int numElems) {
//	double alpha = -1.0;
//	__gpu_matplus_helper(y, x1, x2, numElems, alpha);
//}

/**
 * Implements X = X * alpa (scalar)
 * @param x         address of x2
 * @param alpha     scalar value
 * @param numElems  number of elements
 */
void _gpu_selfmultiply(double* p_x, const double alpha, int numElems) {
	double *x = p_x;
	CUBLAS_CHECK(hipblasDscal(cublasHandle, numElems, &alpha, x, 1));
	CUDA_CHECK(hipDeviceSynchronize());
}

/**
 * implements y = x1 * x2 (contraction) 
 * @param y         address of block y on GPU
 * @param ny        number of indices in y
 * @param yDims     ranges of indices of y
 * @param yInds     labels of y
 * @param x1        address of block x1 on GPU
 * @param n1        number of indices in x1
 * @param x1Dims    ranges of indices of x1
 * @param x1Inds    labels of x1
 * @param x2        address of block x2 on GPU
 * @param n2        number of indices in x2
 * @param x2Dims    ranges of indices of x2
 * @param x2Inds    labels of x2
 */
void _gpu_contract(double* y, int ny, int* yDims, int* yInds,
		double* x1, int n1, int* x1Dims, int* x1Inds,
		double* x2, int n2, int* x2Dims, int *x2Inds) {

//	hipPointerAttribute_t ptrAttr;
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, y));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, x1));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, x2));
//	assert (ptrAttr.memoryType == hipMemoryTypeDevice);


//        int yDims[MAX_RANK], x1Dims[MAX_RANK], x2Dims[MAX_RANK];
//        int i;
//        for(i = 0; i < MAX_RANK; i++) {
//            yDims[i] = nyb[i] - nya[i] + 1;
//            x1Dims[i] = nx1b[i] - nx1a[i] + 1;
//            x2Dims[i] = nx2b[i] - nx2a[i] + 1;
//        }
/************************************************************************/
//printf("X1\n");
//printGPUArray(x1, 10);
//printf("X2\n");
//printGPUArray(x2, 10);
/************************************************************************/
	__gpu_contract_helper(y, ny, yDims, yInds, x1, n1, x1Dims, x1Inds, x2, n2,
			x2Dims, x2Inds);
}

/**
 * implements y = x1 (permutation / assignment / transpose)
 * implements y = x1 * x2 (contraction)
 * @param y         address of block y on GPU
 * @param ny        number of indices in y
 * @param yDims     ranges of indices of y
 * @param yInds     labels of y
 * @param x        address of block x1 on GPU
 * @param nx        number of indices in x1
 * @param xDims    ranges of indices of x1
 * @param xInds    labels of x1
 */
void _gpu_permute(double* y, int ny, int* yDims, int* yInds,
		double* x, int nx, int* xDims, int* xInds) {
//	int yDims[MAX_RANK], x1Dims[MAX_RANK];
//	int i;
//	for (i = 0; i < MAX_RANK; i++) {
//		yDims[i] = nyb[i] - nya[i] + 1;
//		x1Dims[i] = nx1b[i] - nx1a[i] + 1;
//	}
	__gpu_permute_helper(y, ny, yDims, yInds, x, nx, xDims, xInds);
}

/**
 * Mallocs a block of size numElems bytes on GPU
 * @param [in] numElems number of elements
 * @return pointer to allocation
 */
double* _gpu_allocate(int numElems) {

//	std::cout<< "_gpu_allocate called from "<<current_line()<<std::endl;

	double *gpuAddr = NULL;
	//printf ("load_temp : h_addr= %u, h_addr[0] = %lf\n", h_addr, h_addr[0]);
	CUDA_CHECK(hipMalloc((void**)&gpuAddr, (numElems) * sizeof(double)));
	//CUDA_CHECK(hipMemcpy(gpuAddr, h_addr, (*numElems)*sizeof(double), hipMemcpyHostToDevice));
	//*g_addr = gpuAddr;
	CUDA_CHECK(hipMemset((void*)gpuAddr, 0,(numElems) * sizeof(double)));
	CUDA_CHECK(hipDeviceSynchronize());

	assert (gpuAddr != NULL);
//std::cout<<"Allocated on the GPU ..."<<gpuAddr<<std::endl;
//	hipPointerAttribute_t ptrAttr;
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, (void*)gpuAddr));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);
//	CUDA_CHECK(hipDeviceSynchronize());

	//printf("load_temp : gpuAddr=%u, *g_addr=%u\n", gpuAddr, *g_addr);
	return gpuAddr;
}

/**
 * Copies block on GPU back to CPU
 * @param [in]  h_adr address of block on host (Already allocated on CPU)
 * @param [in]  g_adr address of block on gpu
 * @param [in]  numElems number of elements
 */
void _gpu_device_to_host(double* h_addr, double* g_addr, int numElems) {
	double *gpuAddr = g_addr;

//	hipPointerAttribute_t ptrAttr;
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, g_addr));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);
	//printf("unload : gpuAddr=%u, *g_addr=%u\n", gpuAddr, *g_addr);
	//printf("\nunloading gpuaddr :\n");
	//printArray(gpuAddr, 10);
	//printf ("unload : h_addr= %u, h_addr[0] = %lf\n", h_addr, h_addr[0]);
	//CUDA_CHECK(hipMemcpy(gpuAddr, h_addr, (*numElems)*sizeof(double), hipMemcpyHostToDevice));
//	CUDA_CHECK(
			hipMemcpy(h_addr, gpuAddr, (numElems) * sizeof(double),
					hipMemcpyDeviceToHost);
	//CUDA_CHECK(hipFree(gpuAddr)); //$$$$$ TODO GET RID OF THIS $$$$$
//	CUDA_CHECK(hipDeviceSynchronize());
}

/**
 * Copies a block of size numElems * sizeof(double)
 * bytes from Host to GPU
 * @param [in]  h_adr address of block on host
 * @param [in]  g_adr address of block on gpu
 * @param [in]  numElems number of elements
 */
void _gpu_host_to_device(double* h_addr, double* g_addr, int numElems) {

//	hipPointerAttribute_t ptrAttr;
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, g_addr));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);

	//double *gpuAddr = NULL;
	//printf ("load_input : h_addr= %u, h_addr[0] = %lf\n", h_addr, h_addr[0]);
	//CUDA_CHECK(hipMalloc(&gpuAddr, (numElems)*sizeof(double)));
	CUDA_CHECK(hipMemcpy(g_addr, h_addr, (numElems) * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipDeviceSynchronize());
	//*g_addr = gpuAddr;
	//printf("load_input : gpuAddr=%u, *g_addr=%u\n", gpuAddr, *g_addr);
/***********************************************************************************************/
//printf("h_addr before copy");
//printHostArray(h_addr, 10);
//printf("g_addr after copy");
//printGPUArray(g_addr, 10);
/***********************************************************************************************/
}

/**
 * Copies a block of size numElems * sizeof(double)
 * bytes from a memory location dst to src, both on device
 * @param [in]  dst address of destination on device
 * @param [in]  src address of source on device
 * @param [in]  numElems number of elements
 */
void _gpu_device_to_device(double* dst, double *src, int numElems){
	CUDA_CHECK(hipMemcpy(dst, src, (numElems) * sizeof(double), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipDeviceSynchronize());
}


/**
 * Frees block on the GPU
 * @param [in] block to be freed
 */
void _gpu_free(double* g_addr) {
	//double *gpuAddr = g_addr;

//	hipPointerAttribute_t ptrAttr;
//	CUDA_CHECK(hipPointerGetAttributes(&ptrAttr, g_addr));
//	assert(ptrAttr.memoryType == hipMemoryTypeDevice);

	//printf("unload : gpuAddr=%u, *g_addr=%u\n", gpuAddr, *g_addr);
//	CUDA_CHECK(
	hipFree(g_addr);
//	CUDA_CHECK(hipDeviceSynchronize());
}

/**
 * Sets the value of all the doubles int a double block
 */
__global__ void doubleMemSet(double * x, double value, size_t count )
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    for(int i=tid; i<count; i+=stride) {
        x[i] = value;
    }
}

void _gpu_double_memset(double * g_addr, double value, int numElems){
	doubleMemSet<<<REORDER_BLOCKS, REORDER_THREADS>>>(g_addr, value, numElems);
	CUDA_CHECK(hipGetLastError());
}



__global__ void reorderScatter(double* newX, double* oldX, int ndims,
		int size) {
	int blockstep = gridDim.x * blockDim.x;
	int oldIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int newIndex;
	int t;
	int i;

	while (oldIndex < size) {
		t = oldIndex;

		newIndex = t % dimsDev[0] * stepsDev[0];
		t /= dimsDev[0];

		for (i = 1; i < ndims; i++) {
			newIndex += t % dimsDev[i] * stepsDev[i];
			t /= dimsDev[i];
		}

		newX[newIndex] = oldX[oldIndex];
		oldIndex += blockstep;
	}
}

__global__ void reorderGather(double* newX, double* oldX, int ndims, int size) {
	int blockstep = gridDim.x * blockDim.x;
	int newIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int oldIndex;
	int t;
	int i;

	while (newIndex < size) {
		t = newIndex;
		oldIndex = t % dimsDev[0] * stepsDev[0];
		t /= dimsDev[0];

		for (i = 1; i < ndims; i++) {
			oldIndex += t % dimsDev[i] * stepsDev[i];
			t /= dimsDev[i];
		}

		newX[newIndex] = oldX[oldIndex];
		newIndex += blockstep;
	}
}

void printGPUArray(double* g_addr, int size) {
	double * h_addr = (double*) malloc(size * sizeof(double));
	CUDA_CHECK(hipMemcpy(h_addr, g_addr, size * sizeof(double), hipMemcpyDeviceToHost));
	printf("\n");
	for (int i = 0; i < size; i++)
		printf("g[%d]=%lf  ", i, h_addr[i]);
	printf("\n");
}

void printHostArray(double* h_addr, int size) {
	printf("\n");
	for (int i = 0; i < size; i++)
		printf("h[%d]=%lf  ", i, h_addr[i]);
	printf("\n");
}

void __gpu_contract_helper(double* p_y, int ny, int* yDims, int* yInds,
		double* p_x1, int n1, int* x1Dims, int* x1Inds, double* p_x2, int n2,
		int* x2Dims, int* x2Inds) {
	double *y = p_y;
	double *x1 = p_x1;
	double *x2 = p_x2;

	double* scratch1;
	double* scratch2;
	double* scratch3;

	/////////////// Ravi: Dead code?
    // Determine number of elements in the largest of the 3 blocks
    int max_elems = 0;
    int elems = 1;
    for (int i=0; i<n1; i++)
        elems *= x1Dims[i];
    if (elems > max_elems)
        max_elems = elems;
    elems = 0;
    for (int i=0; i<n2; i++)
        elems *= x2Dims[i];
    if (elems > max_elems)
        max_elems = elems;
    elems = 0;
    for (int i=0; i<ny; i++)
        elems *= yDims[i];
    if (elems > max_elems)
        max_elems = elems;
	//////////////////////

	CUDA_CHECK(hipMalloc(&scratch1, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024));
	CUDA_CHECK(hipMalloc(&scratch2, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024));
	CUDA_CHECK(hipMalloc(&scratch3, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024));

    //long BUFF_SIZE = max_elems * sizeof(double); 
    //std::cout<<"Allocating " << BUFF_SIZE << " Memory on GPU for Scratch Blocks" <<std::endl;
	//CUDA_CHECK(hipMalloc(&scratch1, BUFF_SIZE));
	//CUDA_CHECK(hipMalloc(&scratch2, BUFF_SIZE));
	//CUDA_CHECK(hipMalloc(&scratch3, BUFF_SIZE));

	//printf ("p_y=%u p_x1=%u, p_x2=%u\n", p_y, p_x1, p_x2);
	//printf ("y=%u x1=%u x2=%u \n", y, x1, x2);

	int steps[MAX_RANK];
	int yIndsP[MAX_RANK], yDimsP[MAX_RANK];
	int x1IndsP[MAX_RANK], x1DimsP[MAX_RANK];
	int x2IndsP[MAX_RANK], x2DimsP[MAX_RANK];
	int step;
	int lda, ldb;
	int i, j;
	int c, k;
	int size;
	int nc = (n1 + n2 - ny) / 2;
	bool isContractedIndex;

	// determine permutations of x1, x2, and y
	c = 0;
	k = 0;
	for (i = 0; i < n1; i++) {
		isContractedIndex = false;

		for (j = 0; j < n2; j++) {
			if (x1Inds[i] == x2Inds[j]) {
				isContractedIndex = true;
				x1IndsP[n1 - nc + c] = x1Inds[i];
				x1DimsP[n1 - nc + c] = x1Dims[i];
				x2IndsP[c] = x2Inds[j];
				x2DimsP[c] = x2Dims[j];
				c++;
				break;
			}
		}

		if (!isContractedIndex) {
			x1IndsP[k] = x1Inds[i];
			x1DimsP[k] = x1Dims[i];
			yIndsP[k] = x1Inds[i];
			yDimsP[k] = x1Dims[i];
			k++;
		}
	}

	c = 0;
	for (i = 0; i < n2; i++) {
		for (j = 0; j < ny; j++) {
			if (x2Inds[i] == yInds[j]) {
				x2IndsP[nc + c] = x2Inds[i];
				x2DimsP[nc + c] = x2Dims[i];
				yIndsP[k] = yInds[j];
				yDimsP[k] = yDims[j];
				k++;
				c++;
			}
		}
	}

	// copy x1 into scratch3 and then reorder into scratch1
	step = 1;
	for (i = 0; i < n1; i++) {
		for (j = 0; j < n1; j++)
			if (x1Inds[j] == x1IndsP[i]) {
				steps[j] = step;
				break;
			}
		step *= x1DimsP[i];
	}
	size = step;

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), x1Dims, sizeof(int) * n1));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(int) * n1));

	CUDA_CHECK(hipMemcpy(scratch3, x1, size * sizeof(double),hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipDeviceSynchronize());
reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch1, scratch3, n1, size);

    	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipGetLastError());

///***********************************************************************************************/
//printf("scratch1 after reorder\n");
//printGPUArray(scratch1, 10);
///***********************************************************************************************/
	// copy x2 into scratch3 and then reorder into scratch2
	step = 1;
	for (i = 0; i < n2; i++) {
		for (j = 0; j < n2; j++)
			if (x2Inds[j] == x2IndsP[i]) {
				steps[j] = step;
				break;
			}
		step *= x2DimsP[i];
	}
	size = step;

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), x2Dims, sizeof(int) * n2));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(int) * n2));

	CUDA_CHECK(
			hipMemcpy(scratch3, x2, size * sizeof(double),
					hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipDeviceSynchronize());
reorderScatter<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch2, scratch3, n2, size);

    	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
///***********************************************************************************************/
//printf("scratch2 after reorder\n");
//printGPUArray(scratch2, 10);
///***********************************************************************************************/
	// dGemm scratch1 and scratch2 into scratch 3
	double alpha = 1.0;
	double beta = 0.0;

	lda = 1;
	for (i = 0; i < n1 - nc; i++)
		lda *= x1DimsP[i];

	ldb = 1;
	for (i = 0; i < nc; i++)
		ldb *= x2DimsP[i];

	//printf ("Now doing hipblasDgemm\n");
	CUBLAS_CHECK(
			hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, lda, size / ldb, ldb, &alpha, scratch1, lda, scratch2, ldb, &beta, scratch3, lda));

	CUDA_CHECK(hipDeviceSynchronize());
///***********************************************************************************************/
//printf("scratch3 after dgemm\n");
//printGPUArray(scratch3, 10);
///***********************************************************************************************/
	//printf ("Now doing hipblasDcopy\n");
	//CUBLAS_CHECK(hipblasDcopy(cublasHandle, 100, scratch1, 1, scratch3, 1));

	CUDA_CHECK(hipDeviceSynchronize());
	CUBLAS_CHECK(
			hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, lda, size / ldb, ldb, &alpha, scratch1, lda, scratch2, ldb, &beta, scratch3, lda));
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipGetLastError());

	// reorder y from scratch3 to scratch1 and copy back from GPU
	step = 1;
	for (i = 0; i < ny; i++) {
		for (j = 0; j < ny; j++)
			if (yInds[j] == yIndsP[i]) {
				steps[j] = step;
				break;
			}
		step *= yDimsP[i];
	}
	size = step;

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), yDims, sizeof(int) * ny));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(int) * ny));

reorderGather<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch1, scratch3, ny, size);
    	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
	//printf ("y=%u x1=%u x2=%u \n", y, x1, x2);
	CUDA_CHECK(
			hipMemcpy(y, scratch1, size * sizeof(double),
					hipMemcpyDeviceToDevice));

//printf("y after reorder\n");
//printArray(y, 10);

	//printf ("end y=%u x1=%u x2=%u \n", y, x1, x2);

	CUDA_CHECK(hipFree(scratch1));
	CUDA_CHECK(hipFree(scratch2));
	CUDA_CHECK(hipFree(scratch3));

	CUDA_CHECK(hipDeviceSynchronize());

}

void __gpu_permute_helper(double* p_y, int ny, int* yDims, int* yInds,
		double* p_x1, int n1, int* x1Dims, int* x1Inds) {
	double *y = p_y;
	double *x1 = p_x1;

	double* scratch1;
	double* scratch3;

    // Determine number of elements in the larger of the 2 blocks
    int max_elems = 0;
    int elems = 1;
    for (int i=0; i<n1; i++)
        elems *= x1Dims[i];
    if (elems > max_elems)
        max_elems = elems;
    elems = 0;
    for (int i=0; i<ny; i++)
        elems *= yDims[i];
    if (elems > max_elems)
        max_elems = elems;



	CUDA_CHECK(hipMalloc(&scratch1, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024)); 
	CUDA_CHECK(hipMalloc(&scratch3, SCRATCH_BUFFER_SIZE_MB * 1024 * 1024)); 
    //int BUFF_SIZE = max_elems * sizeof(double); 
	//CUDA_CHECK(hipMalloc(&scratch1, BUFF_SIZE));
	//CUDA_CHECK(hipMalloc(&scratch3, BUFF_SIZE));

	//printf ("permute p_y=%u p_x1=%u \n", p_y, p_x1);
	//printf ("permute y=%u x1=%u\n", y, x1);

	int steps[MAX_RANK];
	int yIndsP[MAX_RANK], yDimsP[MAX_RANK];
	int x1IndsP[MAX_RANK], x1DimsP[MAX_RANK];
	int step;
	int i, j;
	int c, k;
	int size;
	bool isContractedIndex;

	c = 0;
	k = 0;
	for (i = 0; i < n1; i++) {

		x1IndsP[i] = x1Inds[i];
		x1DimsP[i] = x1Dims[i];
		yIndsP[i] = x1Inds[i];
		yDimsP[i] = x1Dims[i];
	}

	// copy x1 into scratch3
	step = 1;
	for (i = 0; i < n1; i++) {
		for (j = 0; j < n1; j++)
			if (x1Inds[j] == x1IndsP[i]) {
				steps[j] = step;
				break;
			}
		step *= x1DimsP[i];
	}
	size = step;

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), x1Dims, sizeof(int) * n1));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(int) * n1));

	CUDA_CHECK(hipMemcpy(scratch3, x1, size * sizeof(double), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipDeviceSynchronize());

	// copy scratch1 into scratch3
	step = 1;
	for (i = 0; i < ny; i++) {
		for (j = 0; j < ny; j++)
			if (yInds[j] == x1IndsP[i]) {
				steps[j] = step;
				break;
			}
		step *= x1DimsP[i];
	}
	size = step;

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dimsDev), yDims, sizeof(int) * ny));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(stepsDev), steps, sizeof(int) * ny));

reorderGather<<<REORDER_BLOCKS, REORDER_THREADS>>>(scratch1, scratch3, ny, size);
    	CUDA_CHECK(hipGetLastError());
	// copy scratch1 into y  and copy back from GPU
	CUDA_CHECK(hipMemcpy(y, scratch1, size * sizeof(double), hipMemcpyDeviceToDevice));

	//printf ("end permute y=%u x1=%u\n", y, x1);
	//printf ("end permute : p_y=%u p_x1=%u \n", p_y, p_x1);

	CUDA_CHECK(hipFree(scratch1));
	CUDA_CHECK(hipFree(scratch3));
	CUDA_CHECK(hipDeviceSynchronize());
}

/**
 * Implements Y = Y + alpha * X.
 * @param y         address of y
 * @param x        	address of x
 * @param alpha 	alpha
 * @param numElems  number of elements
 */
void _gpu_axpy(double *p_y, double *p_x, const double alpha, int numElems){
	double *y = p_y;
	double *x = p_x;
	CUBLAS_CHECK(hipblasDaxpy(cublasHandle, numElems, &alpha, x, 1, y, 1));
	CUDA_CHECK(hipDeviceSynchronize());
}

///**
// * Does Y = alpha * X1 + X2
// * @param y         address of array y on GPU
// * @param x1        address of array x1 on GPU
// * @param x2        address of array x2 on GPU
// * @param numElems  number of elements in y, x1 & x2
// * @param alpha     scalar multiplier
// */
//void __gpu_matplus_helper(double* p_y, double* p_x1, double* p_x2, int numElems,
//		const double alpha) {
//
//	double *y = p_y;
//	double *x1 = p_x1;
//	double *x2 = p_x2;
//	if (y == x1) {
//		CUBLAS_CHECK(hipblasDaxpy(cublasHandle, numElems, &alpha, x2, 1, x1, 1));
//	} else if (y == x2) {
//		CUBLAS_CHECK(hipblasDaxpy(cublasHandle, numElems, &alpha, x1, 1, x2, 1));
//	} else {
//		CUDA_CHECK(
//				hipMemcpy(y, x2, (numElems) * sizeof(double),
//						hipMemcpyDeviceToDevice));
//		CUBLAS_CHECK(hipblasDaxpy(cublasHandle, numElems, &alpha, x1, 1, y, 1));
//		//CUBLAS_CHECK(hipblasDswap(cublasHandle, numElems, x2, 1, y, 1));
//	}
//	CUDA_CHECK(hipDeviceSynchronize());
//}

#endif // __GPU_SUPER_INSTRUCTIONS_CU__

